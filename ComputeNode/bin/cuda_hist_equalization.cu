#include "hip/hip_runtime.h"
#include <cmath>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "rgbhsv.h"

#define CUDA 1  
#define COLORDEPTH 256
#define BLOCKSIZE 0

// C function declaration.
void histogram_equalization(unsigned char *, long, int);

// ------------------- CUDA function decleration for Histogram Equalization -------------------------------
__global__ void cu_pll_hist_eq (unsigned char *rgb_image_g, long pix_num, int *counter_array_g, int block_size) {
	// shared memory for color depth.
	//__shared__ int local_depth_hist[COLORDEPTH];
	int local_depth_hist[COLORDEPTH];
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	// working on blocks of some size and calculating histogram value for
	// particular size only. Later we will add up these values. 
	for(int j=0; j<COLORDEPTH; j++){
		local_depth_hist[j] = 0;
	}

	for(int k=tid*block_size; k<(tid+1)*block_size && k<pix_num; k++) {
		local_depth_hist[rgb_image_g[k]]++;
	}

	__syncthreads();

	for(int i=0; i<COLORDEPTH; i++)
		atomicAdd(&(counter_array_g[i]), (local_depth_hist[i])); 
	
	__syncthreads();
}

__global__ void histogram_transform (int * counter_array_g, int *transformed_histogram, int pixel_num){
	//int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	int cumulative_sum = 0;
	for(int i=0;i<tid+1; i++)
		cumulative_sum += counter_array_g[i]; 

	transformed_histogram[tid] = round((double)cumulative_sum/(double)pixel_num * COLORDEPTH);
}

__global__ void transform_image(int *transformed_histogram, unsigned char *final_image, unsigned char *rgb_image_g, int pixel_num){
	int tid = blockDim.x*blockIdx.x + threadIdx.x;

	if(tid < pixel_num)
		final_image[tid] = transformed_histogram[rgb_image_g[tid]] % 256;
}

// --------------- The main function --------------------------- 
int main(int argc, char *argv[]){
	printf("Input parameters:\n%s\n%s\n%s\n\n", argv[0], argv[1], argv[2]);
	int width, height, channel_num;

	// read the image
	unsigned char *rgb_image = stbi_load(argv[1], &width, &height, &channel_num, 0);

	long pixel_num = width * height;
	printf("Width:%d, Height:%d, Channel_num:%d\n\n", width, height, channel_num);

	if (channel_num > 2) {
		hsv *hsv_image;
		hsv_image = (hsv*) malloc (pixel_num*sizeof(hsv));
		unsigned char *brightness_image;
		brightness_image = (unsigned char *) malloc (pixel_num*sizeof(unsigned char));
		
		for (long i=0; i<pixel_num; i++) {
			rgb pixelRGB = {(double)rgb_image[i*channel_num] / 255, (double)rgb_image[i*channel_num+1] / 255, (double)rgb_image[i*channel_num+2] / 255};
			hsv_image[i] = rgb2hsv(pixelRGB);
			brightness_image[i] = round(hsv_image[i].v*255);
		}

		
		histogram_equalization(brightness_image, pixel_num, height);
		
		for (long i=0; i<pixel_num; i++) {
			hsv_image[i].v = (double)brightness_image[i] / 255;
			rgb pixelRGB = hsv2rgb(hsv_image[i]);
			rgb_image[i*channel_num] = pixelRGB.r * 255;
			rgb_image[i*channel_num+1] = pixelRGB.g * 255;
			rgb_image[i*channel_num+2] = pixelRGB.b * 255;
		}
		free(hsv_image);
		free(brightness_image);
	} else if (channel_num == 2) {
		unsigned char *brightness_image;
		brightness_image = (unsigned char *) malloc (pixel_num*sizeof(unsigned char));
		for (long i=0; i<pixel_num; i++) {
			brightness_image[i] = rgb_image[i*2];
		}

		histogram_equalization(brightness_image, pixel_num, height);

		for (long i=0; i<pixel_num; i++) {
			rgb_image[i*2] = brightness_image[i];
		}
		free(brightness_image);
	} else {
		histogram_equalization(rgb_image, pixel_num, height);
	}
	
	// write the image
	if (argc == 4 && strcmp(argv[3], "png")==0){
		stbi_write_png(argv[2], width, height, channel_num, rgb_image, width*channel_num);
	} else {
		stbi_write_jpg(argv[2], width, height, channel_num, rgb_image, 100);
	}
	printf("Processing is finished!\n");

	stbi_image_free(rgb_image);

	return 0;
}

void histogram_equalization(unsigned char *image, long pixel_num, int height) {
	// initialize histogram array and transform array
	if (!CUDA) {
		struct timespec start, stop; 
		double time;
		unsigned char *imgout = (unsigned char *)malloc(pixel_num);

		int counter_array[COLORDEPTH], trans_table[COLORDEPTH];
		
		if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}
			
		for (int i=0; i<COLORDEPTH; i++) {
				counter_array[i] = 0;
		}
		
		// calculate histogram
		for (int i=0; i<pixel_num; i++) {
			counter_array[image[i]]++;
		}
		
		// build transform function
		long frequency_sum = 0;
		for (int i=0; i<COLORDEPTH; i++) {
			frequency_sum += counter_array[i];
			trans_table[i] = round((double)frequency_sum / (double)pixel_num * COLORDEPTH);
		}

		// transform image
		for (int i=0; i<pixel_num; i++) {
			imgout[i] = trans_table[image[i]];
		}
		if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror("clock gettime");}   
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;

		printf("Execution time baseline is = %f sec\n", time);
	}
	else if(CUDA){
		// Timing structures
		struct timespec start, stop; 
		double time;
	
		int block_size = BLOCKSIZE ? height:1;
		unsigned char *rgb_image_g;
		unsigned char *final_image;
		int *counter_array_g;
		int *transformed_histogram;
	
		// declare memory in GPU.
		hipMalloc((void **)&rgb_image_g, sizeof(unsigned char)*pixel_num);
		hipMalloc((void **)&final_image, sizeof(unsigned char)*pixel_num);
		hipMalloc((void **)&counter_array_g, sizeof(int)*COLORDEPTH);
		hipMalloc((void **)&transformed_histogram, sizeof(int)*COLORDEPTH);

		// Transfer image from host to device
		hipMemcpy(rgb_image_g, image, sizeof(unsigned char)*pixel_num, hipMemcpyHostToDevice);
	
		dim3 block(block_size,1);
		dim3 grid(((pixel_num + block_size - 1)/block_size),1);
	
		if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}
		
		// calling kernal to do histogram eq.
		cu_pll_hist_eq<<<grid,block>>>(rgb_image_g, pixel_num, counter_array_g, block_size);
	
		// calling transformation function.
		histogram_transform<<<1, COLORDEPTH>>>(counter_array_g, transformed_histogram, pixel_num);
	
		// calling image transformation function.
		transform_image<<<grid, block>>>(transformed_histogram, final_image, rgb_image_g, pixel_num);

		if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror("clock gettime");}   
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;

		printf("Cuda Execution time = %f sec\n", time);
		printf("================================\n");

		// retrieving image from device
		//unsigned char *final_image_host = (unsigned char *)malloc(sizeof(unsigned char)*pixel_num);
		hipMemcpy(image, final_image, sizeof(unsigned char)*pixel_num, hipMemcpyDeviceToHost);
		
		hipFree(rgb_image_g);
		hipFree(final_image);;
		hipFree(&counter_array_g);
		hipFree(&transformed_histogram);
	}
}
